//
// Created by djang on 2021-01-09.
//

#include "util.h"

#include <hip/hip_runtime.h>
#include <iostream>

void cuda_validate(hipError_t err, const char *file, int line) {
    if (err) {
        std::cerr << "CUDA: " << file << "(" << line << "): "
                  << "error: " << hipGetErrorString(err) << " : return code '0x" << std::hex << err << std::dec
                  << "'\n";
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }
}
